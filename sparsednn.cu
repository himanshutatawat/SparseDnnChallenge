#include "hip/hip_runtime.h"
%%cu
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <boost/algorithm/string.hpp>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#define row 60000
#define column 1024

using namespace std;
using namespace boost::algorithm;

__global__ void ReluActivation(float * MNIST,float* layer,float * Out)
{
    
    int Row=blockIdx.y*blockDim.y+threadIdx.y;
    int Column=blockIdx.x*blockDim.x+threadIdx.x;
    if(Row<60000 && Column < 1024)
    {
        float bias=0;
        for(int z=0;z<1024;z++)   bias+=MNIST[Row*1024+z]*layer[z*1024+Column];
        bias = bias - 0.3;
     
        if((bias)<0)  bias = 0;
        if((bias) >32)  bias = 32;
        Out[Row*1024+Column]=bias;
    }
    __syncthreads();
}
int main() {

    ifstream file("drive/MyDrive/sparse-images-1024.tsv");
    string line;
    int counter = 0;
    float *Y_zero;
    Y_zero  = (float*)malloc(60000*1024*sizeof(float));
    for(int i=0; i<row*column; i++)    Y_zero[i]=0;
    
    while (getline(file, line)) {
        counter++;
        vector<string> parts;
        split(parts, line, boost::is_any_of("\t"));
        string st1 = parts[0];
        int r = stoi(st1);
        string st2 = parts[1];
        int c = stoi(st2);

        Y_zero[(r-1)*1024+(c-1)]=1;
    }
    file.close();

    printf("TNZC: %d ", counter);

    for(int i=1; i<=120; i++){
      string  nlayer="n1024-l";
      string lay=to_string(i);
      nlayer="drive/MyDrive/neuron1024/"+nlayer+lay+".tsv";
      ifstream files(nlayer);
      string line;
      float *weight;
      weight=(float*)malloc(1024*1024*sizeof(float));
      for(int i=0; i<1024*1024; i++)     weight[i]=0;


    while (getline(files, line)) {
        vector<string> parts;
        split(parts, line, boost::is_any_of("\t"));
        // TODO Your code goes here!
        string st1 = parts[0];
        int ro = stoi(st1);
        string st2 = parts[1];
         int co = stoi(st2);
        weight[(ro-1)*1024+(co-1)]=0.0625;
    }
  
    files.close();

    float *D_MNIST;
    hipMalloc(&D_MNIST,60000*1024*sizeof(float));
    float *D_Weight;
    hipMalloc(&D_Weight,1024*1024*sizeof(float));
    float* D_out;
    hipMalloc(&D_out,60000*1024*sizeof(float)); 
    hipMemcpy(D_MNIST,Y_zero,60000*1024*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(D_Weight,weight,1024*1024*sizeof(float),hipMemcpyHostToDevice);

    dim3 dimGrid(32, 1875);
    dim3 dimBlock(32, 32);
    /* Start measuring time
    struct timeval begin, end;
    gettimeofday(&begin, 0);
    */

    ReluActivation<<<dimGrid, dimBlock>>>(D_MNIST, D_Weight, D_out);
    hipDeviceSynchronize(); 

   /* Stop measuring time and calculate the elapsed time
    gettimeofday(&end, 0);
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
    double elapsed = seconds + microseconds*1e-6;
    
    printf("Time measured: %.6f seconds.\n", elapsed);


*/
      hipMemcpy(Y_zero,D_out,60000*1024*sizeof(float),hipMemcpyDeviceToHost);
          int g=0;
    for(int i=0;i<1024;i++){
        for(int j=0;j<60000;j++){
            if(Y_zero[i*60000+j]!=0){
                g++;        //calculating the non- zero elements
            }
        }
    }
    printf("%d ",g);
}
int active_images=0;
  for(int i=0;i<60000;i++)
  {
      for(int j=0;j<1024;j++)
      {
          if(Y_zero[i*1024+j]!=0)
          {
          printf("%d\n ",i+1);
           active_images++;
            break;
          }
      }
  }
    printf("\nTotal Active images %d           ",active_images);
    return 0;
}